
#include <hip/hip_runtime.h>
#include<iostream>
#include<bits/stdc++.h>
#include<vector>
#include<queue>
#include<fstream>
#include<set>
 using namespace std;
int *offset,*edges,*distArray,offCount,eCount;
__global__ void updateQueue(int* d_offset,int* d_edges,int * d_distArray,int* d_currentQueue,int* d_nextQueue,int *currentSize,int *nextSize){
    int i;
    int tid=(blockDim.x*blockIdx.x)+threadIdx.x;
    int k=currentSize[0];
    if(tid<k){
        printf("%d ",d_currentQueue[tid]);
        int temp1=d_offset[d_currentQueue[tid]];
        int temp2=d_offset[d_currentQueue[tid]+1];
        for(i=temp1;i<temp2;i++){
            int b = d_edges[i];
            if(d_distArray[b]==-1){
                d_distArray[b]=d_distArray[d_currentQueue[tid]]+1;
                int position = atomicAdd(nextSize, 1);
				        d_nextQueue[position] = b;
            }
        }
    }
    
}

void BFS(int s){
    int *d_offset,*d_edges,*d_distArray,*d_currentQueue,*d_nextQueue,*d_currentSize,*d_nextSize;//This is for GPU
    int threads=1<<10;
    int size = sizeof(int)*(offCount-1);//This is for CPU
    int *Queue = new int[offCount-1];//This is for CPU
    int *Qsize=new int[1];//This is for CPU
    hipMalloc(&d_offset,sizeof(int)*offCount);
    hipMalloc(&d_edges,sizeof(int)*eCount);
    hipMalloc(&d_distArray,size);
    hipMalloc(&d_currentQueue,size);
    hipMalloc(&d_nextQueue,size);
    hipMalloc(&d_currentSize,sizeof(int));
    hipMalloc(&d_nextSize,sizeof(int));

    distArray[s]=0;
    hipMemcpy(d_offset,offset,sizeof(int)*offCount,hipMemcpyHostToDevice);
    hipMemcpy(d_edges,edges,sizeof(int)*eCount,hipMemcpyHostToDevice);
    hipMemcpy(d_distArray,distArray,size,hipMemcpyHostToDevice);
    int blocks = (offCount+threads-2)/(offCount-1);
    Queue[0]=s;
    Qsize[0]=1;
    int *x=new int[1];
    x[0]=0;
    cout<<"Source as: "<<s<<": ";
    
    while(Qsize[0]>0){
        set<int> temp;
        int i;
        hipMemcpy(d_nextSize,x,sizeof(int),hipMemcpyHostToDevice);
        hipMemcpy(d_currentQueue,Queue,size,hipMemcpyHostToDevice);
        hipMemcpy(d_currentSize,Qsize,sizeof(int),hipMemcpyHostToDevice);
        updateQueue<<<blocks,threads>>>(d_offset,d_edges,d_distArray,d_currentQueue,d_nextQueue,d_currentSize,d_nextSize);
        hipMemcpy(Qsize,d_nextSize,sizeof(int),hipMemcpyDeviceToHost);
        hipMemcpy(Queue,d_nextQueue,size,hipMemcpyDeviceToHost);


        for(i=0;i<*Qsize;i++){
          temp.insert(Queue[i]);
        }
        int y=temp.size();
        i=0;
        for (auto itr : temp)
        {
          Queue[i]=itr;
          i++;
        } 
        Qsize=&y;

    }
    hipMemcpy(distArray,d_distArray,size,hipMemcpyDeviceToHost);
}

void reset(int n){
    int i;
    for(i=0;i<n;i++){
        distArray[i]=-1;
    }
}
int main(int argc, char* argv[]){
    int source,qCount;
    cin>>offCount;
    cin>>eCount;
    cin>>qCount;
    int temp = (int)ceil((offCount-1)/8);
    temp++;
    cout<<offCount-1<<" "<<eCount<<" "<<endl;
    offset=new int[offCount];
    edges=new int[eCount];
    distArray=new int[offCount-1];
    int i;
    for(i=0;i<offCount;i++){
        cin>>offset[i];
    }
    for(i=0;i<eCount;i++){
        cin>>edges[i];
    }
    cout<<"Graph read successfully"<<endl;
    for(i=0;i<qCount;i++){
        cin>>source;
        reset(offCount-1);
        BFS(source);
        cout<<endl;
    }
    return 0;
    
}