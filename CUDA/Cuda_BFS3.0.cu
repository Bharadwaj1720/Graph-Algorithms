
#include <hip/hip_runtime.h>
#include<iostream>
#include<bits/stdc++.h>
#include<vector>
#include<queue>
#include<fstream>
 using namespace std;
int *offset,*edges,*distArray,offCount,eCount;
__global__ void updateQueue(int* d_offset,int* d_edges,int * d_distArray,int *d_count,int* d_level,int* d_offCount){
    int i;
    int tid=(blockDim.x*blockIdx.x)+threadIdx.x;
    int k=(d_offCount[0]) -1;
    if(tid<k && d_level[0]==d_distArray[tid]){
        d_count[0]++;
        printf("%d ",tid);
        int temp1=d_offset[tid];
        int temp2=d_offset[tid+1];
        for(i=temp1;i<temp2;i++){
            int b = d_edges[i];
            if(d_distArray[b]==-1){
                d_distArray[b]=d_distArray[tid]+1;
            }
        }
    }
    
}

void BFS(int s){
    int *d_offset,*d_edges,*d_distArray,*d_count, *d_offCount,*d_level;//This is for GPU
    int threads=1<<10;
    int *count=new int[1];
    int *level=new int [1];
    count[0]=1;
    level[0]=0;
    int size = sizeof(int)*(offCount-1);//This is for CPU
    hipMalloc(&d_offset,sizeof(int)*offCount);
    hipMalloc(&d_edges,sizeof(int)*eCount);
    hipMalloc(&d_distArray,size);
    hipMalloc(&d_level,sizeof(int));
    hipMalloc(&d_count,sizeof(int));
    hipMalloc(&d_offCount,sizeof(int));
    distArray[s]=0;
    hipMemcpy(d_offset,offset,sizeof(int)*offCount,hipMemcpyHostToDevice);
    hipMemcpy(d_edges,edges,sizeof(int)*eCount,hipMemcpyHostToDevice);
    hipMemcpy(d_distArray,distArray,size,hipMemcpyHostToDevice);
    hipMemcpy(d_offCount,&offCount,sizeof(int),hipMemcpyHostToDevice);
    int blocks = (offCount+threads-2)/(offCount-1);
    int *x=new int[1];
    x[0]=0;
    cout<<"Source as: "<<s<<": ";
    hipMemcpy(level,x,sizeof(int),hipMemcpyHostToDevice);
    while(count[0]!=0){
        hipMemcpy(d_count,x,sizeof(int),hipMemcpyHostToDevice);
        hipMemcpy(d_level,level,sizeof(int),hipMemcpyHostToDevice);
        updateQueue<<<blocks,threads>>>(d_offset,d_edges,d_distArray,d_count,d_level,d_offCount);
        hipMemcpy(count,d_count,sizeof(int),hipMemcpyDeviceToHost);
        hipMemcpy(level,d_level,sizeof(int),hipMemcpyDeviceToHost);
        level[0]++;
    }
    hipMemcpy(distArray,d_distArray,size,hipMemcpyDeviceToHost);
}

void reset(int n){
    int i;
    for(i=0;i<n;i++){
        distArray[i]=-1;
    }
}
int main(int argc, char* argv[]){
    int source,qCount;
    cin>>offCount;
    cin>>eCount;
    cin>>qCount;
    int temp = (int)ceil((offCount-1)/8);
    temp++;
    cout<<offCount-1<<" "<<eCount<<" "<<endl;
    offset=new int[offCount];
    edges=new int[eCount];
    distArray=new int[offCount-1];
    int i;
    for(i=0;i<offCount;i++){
        cin>>offset[i];
    }
    for(i=0;i<eCount;i++){
        cin>>edges[i];
    }
    cout<<"Graph read successfully"<<endl;
    for(i=0;i<qCount;i++){
        cin>>source;
        reset(offCount-1);
        BFS(source);
        cout<<endl;
    }
    return 0;
    
}